#include "hip/hip_runtime.h"
#include "header.h"
#define MAX_BLOCK_THREADS 1024


__global__ void addKernel(double *dev_a, double *dev_b, double *dev_c, int n, int dim, double t)
{
    int i = threadIdx.x;
	int b = blockIdx.x;
	int d = blockDim.x;

	int idx = b * d + i;

	if (idx < n * dim) 
		dev_c[idx] = dev_a[idx] + dev_b[idx] * t;
}

// Helper function for using CUDA to set points in parallel.
hipError_t setPointsWithCuda(Point *points, int n, int dim, double t, double *dev_vecInitloc, double *dev_vecV, double *dev_vecCurrentloc)
{
	char errorBuffer[100];
	hipError_t cudaStatus;
	int numBlocks;

    // Choose which GPU to run on
    cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, dev_vecInitloc, dev_vecV, dev_vecCurrentloc,
	"hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");

	numBlocks = (n*dim) / MAX_BLOCK_THREADS;
	if ( (n*dim) % MAX_BLOCK_THREADS)
		numBlocks++;

    // Launch a kernel on the GPU 
    addKernel<<<numBlocks, MAX_BLOCK_THREADS >>>(dev_vecInitloc, dev_vecV, dev_vecCurrentloc, n, dim, t);

    // Check for any errors launching the kernel

    cudaStatus = hipGetLastError();
	sprintf(errorBuffer, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	checkError(cudaStatus, dev_vecInitloc, dev_vecV, dev_vecCurrentloc, errorBuffer);
    

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.

    cudaStatus = hipDeviceSynchronize();
	sprintf(errorBuffer, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	checkError(cudaStatus, dev_vecInitloc, dev_vecV, dev_vecCurrentloc, errorBuffer);


	// Copy output vector from GPU buffer to host memory.

	cudaStatus = hipMemcpy(points->vecCurrentloc, dev_vecCurrentloc, dim * sizeof(double) * n , hipMemcpyDeviceToHost);
	checkError(cudaStatus, dev_vecInitloc, dev_vecV, dev_vecCurrentloc, errorBuffer);

    return cudaStatus;
}

hipError_t initCudaMemory(Point *points, int n, int dim, double **dev_a, double **dev_b, double **dev_c)
{
	hipError_t cudaStatus;
	char errorBuffer[100];

	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, *dev_a, *dev_b, *dev_c,
		"hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");

	// Allocate GPU buffers for three vectors 

	cudaStatus = hipMalloc((void**)dev_a, n * dim * sizeof(double));
	checkError(cudaStatus, *dev_a, *dev_b, *dev_c, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)dev_b, n * dim * sizeof(double));
	checkError(cudaStatus, *dev_a, *dev_b, *dev_c, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)dev_c, n * dim * sizeof(double));
	checkError(cudaStatus, *dev_a, *dev_b, *dev_c, "hipMalloc failed!");

	// Copy input vectors from host memory to GPU buffers.

	cudaStatus = hipMemcpy(*dev_a, points->vecInitloc, dim * sizeof(double) * n, hipMemcpyHostToDevice);
	checkError(cudaStatus, *dev_a, *dev_b, *dev_c, "hipMemcpy failed!\n");

	cudaStatus = hipMemcpy(*dev_b, points->vecV, dim * sizeof(double) * n, hipMemcpyHostToDevice);
	checkError(cudaStatus, *dev_a, *dev_b, *dev_c, "hipMemcpy failed!\n");

	cudaStatus = hipGetLastError();
	sprintf(errorBuffer, "initCudaMemory failed: %s\n", hipGetErrorString(cudaStatus));
	checkError(cudaStatus, *dev_a, *dev_b, *dev_c, "hipMemcpy failed!\n");

	return cudaStatus;
}

void freeCudaMemory(double *dev_a, double *dev_b, double *dev_c)
{
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void checkError(hipError_t cudaStatus, double *dev_a, double *dev_b, double *dev_c, const char* errorMessage)
{
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, errorMessage);
		fprintf(stderr, "\n");
		freeCudaMemory(dev_a, dev_b, dev_c);
	}
}

// unused below

__global__ 	void fKernel(int *dev_f, double *dev_w, double *dev_points,int n,int  dim) {
	int i = threadIdx.x;
	int b = blockIdx.x;
	int d = blockDim.x;

	int idx = b * d + i;

	if (idx < n) {
		double res = dev_w[0];
		for (int k = 0; k < dim; k++)
			res += dev_w[k + 1] * dev_points[idx*dim+k];

		dev_f[idx]=(res >= 0) ? A : B;
	}
}

void markWithCuda(Classifier C, double *weights, int *fArray, double *dev_points) {

	int *dev_f;
	double *dev_w;
	int numBlocks;

	// Choose which GPU to run on
	hipSetDevice(0);


	hipMalloc((void**)&dev_f, C.N  * sizeof(int));
	hipMalloc((void**)&dev_w, (C.K+1) * sizeof(double));
	hipMemcpy(dev_f, fArray, sizeof(int) * C.N, hipMemcpyHostToDevice);
	hipMemcpy(dev_w, weights, (C.K+1) * sizeof(double), hipMemcpyHostToDevice);

	numBlocks = (C.N) / MAX_BLOCK_THREADS;
	if ((C.N) % MAX_BLOCK_THREADS)
		numBlocks++;

	// Launch a kernel on the GPU 
	fKernel <<<numBlocks, MAX_BLOCK_THREADS >>> (dev_f, dev_w, dev_points,C.N, C.K);

	hipMemcpy(fArray, dev_f,C.N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_f);
	hipFree(dev_w);
}
